#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#ifndef M_PI
#define M_PI 3.14159265
#endif

#include "cuda_mmult_kernels.h"

// define macro OUTPUT to print input & output matrix
//#define OUTPUT

// define macro QUERY_DEVICES to print device information
//#define QUERY_DEVICES

void checkCUDAError(const char *msg);

void zeroMatrix(float *A, int n);
void dstMatrix(float *A, int n);
void CPU_matrixMult(float *A, float *B, float *C, int n, int repeats);
void CUDA_matrixMult(float *A, float *B, float *C, int n, int repeats);
void CUDA_freeMatrix(float *Ad);
void printMatrix(char* name, float *A, int n);
void printDeviceInfo(hipDeviceProp_t devProp);

int main(int argc, char *argv[]) {
   float *A,*B,*C; /* arrays for matrices */
   int n, m; /* n=matrix size, m=repeats */
   
   hipEvent_t start_timer, stop_timer;
   float gpu_time;
 
#ifdef QUERY_DEVICES
   // Number of CUDA devices
   int devCount;
   hipGetDeviceCount(&devCount);
   printf("CUDA Device Query...\n");
   printf("There are %d CUDA devices.\n", devCount);

   // Iterate through devices
   for (int i = 0; i < devCount; ++i) {
       // Get device properties
       printf("\nCUDA Device #%d\n", i);
       hipDeviceProp_t devProp;
       hipGetDeviceProperties(&devProp, i);
       printDeviceInfo(devProp);
   }
#endif

   if (argc < 3) {
      printf("Error: please specify matrix size and number of multiplications: \n");
      printf("Usage: %s <size> <repeat> \n", argv[0]);
      exit(1);      
   };
   
   /* read matrix size and number of repeats */

   n = atoi(argv[1]);
   m = atoi(argv[2]);

   if (n % TILE_SIZE != 0) {
      printf("Error: matrix size has to be a multiple of tile size %d \n", TILE_SIZE);
      exit(1);      
   };

   hipEventCreate(&start_timer);
   hipEventCreate(&stop_timer);

   printf("Matrix mult. of size %d (%d repeats): \n", n, m);

   /* allocate and initialise matrices in host memory */

   int size = n*n*sizeof(float);

   A = (float *) malloc(size);
   dstMatrix(A,n);
   B = (float *) malloc(size);
   dstMatrix(B,n);
   C = (float *) malloc(size);
   zeroMatrix(C,n);
   
#ifdef OUTPUT
   printMatrix("A",A,n);
   printMatrix("B",B,n); 
#endif

   /* allocate matrices in device memory and transfer matrices from host to device memory */
   
   float *Ad, *Bd, *Cd;
   
   hipMalloc((void**)&Ad, size); checkCUDAError("allocate memory for A");
   hipMalloc((void**)&Bd, size); checkCUDAError("allocate memory for B");
   hipMalloc((void**)&Cd, size); checkCUDAError("allocate memory for C");

   hipMemcpy(Ad,A, size, hipMemcpyHostToDevice); checkCUDAError("memory of A not transferred");
   hipMemcpy(Bd,B, size, hipMemcpyHostToDevice); checkCUDAError("memory of B not transferred");
   hipMemcpy(Cd,C, size, hipMemcpyHostToDevice); checkCUDAError("memory of C not transferred");

   /* perform matrix multiplication (m repeats) */

   hipEventRecord(start_timer, 0);
      //CPU_matrixMult(A, B, C, n, m);
      CUDA_matrixMult(Ad,Bd,Cd,n,m);
   hipEventRecord(stop_timer, 0);

   /* transfer result matrix back from device to host memory and deallocate device matrices */

   hipMemcpy(C,Cd, size, hipMemcpyDeviceToHost); checkCUDAError("memory of C not transferred back");

   hipFree(Ad);
   hipFree(Bd);
   hipFree(Cd);

#ifdef OUTPUT
   printMatrix("C", C, n);
#endif

   /* deallocate host matrices, print results */

   free(A);
   free(B);
   free(C);
   
   hipEventSynchronize(stop_timer);
   hipEventElapsedTime(&gpu_time, start_timer, stop_timer);

   printf("Elapsed time : %.3f s \n", gpu_time / 1000.0f);
   printf("Performance  : %.0f MFlop/s \n", float(m) * (2.0f * n - 1.0f) * n * n / (gpu_time / 1000.0f * 1024.f * 1024.f));

   hipEventDestroy(start_timer);
   hipEventDestroy(stop_timer);

   return(0);
}

/* set Matrix values to zero */
void zeroMatrix(float *A, int n)
{
   int i,k;

   for (i=0; i<n; i++)
     for (k=0; k<n; k++)
	    A[i*n+k] = 0;
}

/* initialise Matrix: discrete Sine Transform */
void dstMatrix(float *A, int n)
{
   int i,k;

   for (i=0; i<n; i++)
     for (k=0; k<n; k++)
	    A[i*n+k] = sin( ((i+1)*(k+1)*M_PI)/(n+1));
}



/* 
 * matrix multiplication C += A*B 
 *  -> standard C implementation
 */
void CPU_matrixMult(float *A, float *B, float *C, int n, int repeats) {
	int i,j,k;
    float tmp;

	for(int r=0; r<repeats; r++) {
    	for (i=0; i<n; i++) {
			for (j=0; j<n; j++) {
				tmp = A[i*n+j];

				for (k=0; k<n; k++) {
					C[i*n+k] += tmp * B[j*n+k];
				}
    		}
		}
    }
}

/* 
 * matrix multiplication C += A*B 
 *  -> CUDA implementation: kernel invocation
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 3)
 */
__host__ void CUDA_matrixMult(float *Ad, float *Bd, float *Cd, int n, int repeats) {
   dim3 dimBlock(TILE_SIZE,TILE_SIZE);
   dim3 dimGrid(n/TILE_SIZE,n/TILE_SIZE);
   
   for(int i=0; i<repeats; i++) {
      // matrixMultKernel_global<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n);
      //  matrixMultKernel_tiled<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n);
      // matrixMultKernel_coalesced<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n);
      matrixMultKernel_overlap<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n);
   }
   checkCUDAError("matrix multiplication kernel failed");
}

/* print Matrix */
void printMatrix(char* name, float *A, int n)
{
   int i,k;

   printf("Matrix %s (size %d)\n",name,n);

   for (i=0; i<n; i++) {
     for (k=0; k<n; k++) {
       printf("%f ", A[i*n+k]);
     }

     printf("\n");
   }
}

/*
 * helper function to check for errors in CUDA calls
 * source: NVIDIA
 */
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();

    if( hipSuccess != err) {
        fprintf(stderr, "\nCuda error (%s): %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}

#ifdef QUERY_DEVICES
// Print device info
void printDeviceInfo(hipDeviceProp_t devProp) {
    printf("Revision number:               %d.%d\n", devProp.major, devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu MB\n",  devProp.totalGlobalMem / (1024 * 1024));
    printf("Total shared memory per block: %lu kB\n",  devProp.sharedMemPerBlock / 1024);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu MB\n",  devProp.memPitch / (1024 * 1024));
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);

    printf("Maximum dimensions of block:   %d %d %d\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
    printf("Maximum dimensions of grid:    %d %d %d\n", devProp.maxGridSize[0], devProp.maxGridSize[1], devProp.maxGridSize[2]);

    printf("Clock rate:                    %d MHz\n",  devProp.clockRate / 1000);
    printf("Total constant memory:         %lu kB\n",  devProp.totalConstMem / 1024);
    printf("Texture alignment:             %lu B\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    printf("\n");
}
#endif
