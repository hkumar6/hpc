#include "hip/hip_runtime.h"
#include "cuda_mmult_kernels.h"

/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 4)
 *  -> Features: none (basic tiled version, using only global memory)
 */
__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n)
{
   int i = blockIdx.x * TILE_SIZE + threadIdx.x;
   int k = blockIdx.y * TILE_SIZE + threadIdx.y;
   
   float Celem = 0;
   
   for(int j=0; j<n; j++) {
      float Aelem = Ad[i*n+j];
      float Belem = Bd[j*n+k];
      Celem += Aelem*Belem;
   }
   
   Cd[i*n+k] += Celem;
}

/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 5)
 *  -> Features:
 *     - tiled matrix multiplication with use of shared memory
 */
__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n)
{
   __shared__ float Ads[TILE_SIZE][TILE_SIZE];
   __shared__ float Bds[TILE_SIZE][TILE_SIZE];

   int tx = threadIdx.x;
   int ty = threadIdx.y;
   
   int i = blockIdx.x * TILE_SIZE + tx;
   int k = blockIdx.y * TILE_SIZE + ty;
   
   float Celem = 0;
   
   for(int m=0; m < n/TILE_SIZE; m++) {
      Ads[tx][ty] = Ad[ i*n + m*TILE_SIZE+ty];
      Bds[tx][ty] = Bd[ (m*TILE_SIZE+tx)*n + k];
      __syncthreads();
      
      for(int j=0; j<TILE_SIZE; j++)
	     Celem += Ads[tx][j]*Bds[j][ty];
   
      __syncthreads();
   };

   Cd[i*n+k] += Celem;
}


/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 5)
 *  -> Features:
 *     - tiled matrix multiplication with use of shared memory
 *     - coalesced memory access
 */
__global__ void matrixMultKernel_coalesced(float* Ad, float* Bd, float* Cd, int n)
{
   __shared__ float Ads[TILE_SIZE][TILE_SIZE];
   __shared__ float Bds[TILE_SIZE][TILE_SIZE];

   int tx = threadIdx.x;
   int ty = threadIdx.y;
   
   int i = blockIdx.y * TILE_SIZE + ty;
   int k = blockIdx.x * TILE_SIZE + tx;
   
   float Celem = 0;
   
   for(int m=0; m < n/TILE_SIZE; m++) {
      Ads[ty][tx] = Ad[ i*n + m*TILE_SIZE+tx];
      Bds[ty][tx] = Bd[ (m*TILE_SIZE+ty)*n + k];
      __syncthreads();
      
      for(int j=0; j<TILE_SIZE; j++)
	     Celem += Ads[ty][j]*Bds[j][tx];
   
      __syncthreads();
   };
   Cd[i*n+k] += Celem;
}


/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 5)
 *  -> Features:
 *     - tiled matrix multiplication with use of shared memory
 *     - coalesced memory access
 *     - overlapping loads of subsequent tile pairs (using registers & shared memory)
 */
__global__ void matrixMultKernel_overlap(float* Ad, float* Bd, float* Cd, int n)
{
   __shared__ float Ads[TILE_SIZE][TILE_SIZE];
   __shared__ float Bds[TILE_SIZE][TILE_SIZE];

   float Adr;
   float Bdr;

   int tx = threadIdx.x;
   int ty = threadIdx.y;
   
   int i = blockIdx.y * TILE_SIZE + ty;
   int k = blockIdx.x * TILE_SIZE + tx;
   
   float Celem = 0;
   int m = 0;

   /* load the first tile into the registers */
   Adr = Ad[ i*n + m*TILE_SIZE+tx];
   Bdr = Bd[ (m*TILE_SIZE+ty)*n + k];

   for(m=1; m < n/TILE_SIZE; m++) {
      /* copy current tile from registers into shared memory */
      Ads[ty][tx] = Adr;
      Bds[ty][tx] = Bdr;
      __syncthreads();

      /* load the next tile into the registers */
      Adr = Ad[ i*n + m*TILE_SIZE+tx];
      Bdr = Bd[ (m*TILE_SIZE+ty)*n + k];

      /* compute from shared memory */
      #pragma unroll
      for(int j=0; j<TILE_SIZE; j++)
	     Celem += Ads[ty][j]*Bds[j][tx];
   
      __syncthreads();
   };

   /* compute final tile from register */
   Ads[ty][tx] = Adr;
   Bds[ty][tx] = Bdr;
   __syncthreads();
   #pragma unroll
   for(int j=0; j<TILE_SIZE; j++)
      Celem += Ads[ty][j]*Bds[j][tx];

   Cd[i*n+k] += Celem;
}
